#include "hip/hip_runtime.h"
﻿#include "hip/hip_runtime.h"
#include ""
#include <stdio.h>
#include <cstdlib>

void init_matrix(int N, int M, int K, double* A, double* B, double* C);
void print_matrix(int N, int M, double* matrix);

#define BLOCK_SIZE 32

__global__ void blas_dgemmCUDAv1(const double* A, const double* B, double* C, int N, int M, int K)
{
    int row = blockIdx.y * BLOCK_SIZE + threadIdx.y;
    int col = blockIdx.x * BLOCK_SIZE + threadIdx.x;
    double sum = 0.0;

    if (row < N && col < K) 
    {
        for (int i = 0; i < M; ++i) 
        {
            sum += A[row * M + i] * B[i * K + col];
        }
        C[row * K + col] = sum;
    }
}

void run_dgemmCUDAv1(const double* d_A, const double* d_B, double* d_C, int N, int M, int K)
{
    /* Set size block (BLOCK_SIZE x BLOCK_SIZE) */
    dim3 dimBlock(BLOCK_SIZE, BLOCK_SIZE);

    /* Each block thread a part of the matrix with the size BLOCK_SIZE x BLOCK_SIZE */
    dim3 dimGrid((K + BLOCK_SIZE - 1) / BLOCK_SIZE, (N + BLOCK_SIZE - 1) / BLOCK_SIZE);

    /* Run CUDA-core */
    blas_dgemmCUDAv1 << <dimGrid, dimBlock >> > (d_A, d_B, d_C, N, M, K);
}

__global__ void blas_dgemmCUDAv2(const double* A, const double* B, double* C, int N, int M, int K) {
    // Размер блоков
    __shared__ double tileA[BLOCK_SIZE][BLOCK_SIZE];
    __shared__ double tileB[BLOCK_SIZE][BLOCK_SIZE];

    // Индексы строки и столбца для матрицы C
    int row = blockIdx.y * BLOCK_SIZE + threadIdx.y;
    int col = blockIdx.x * BLOCK_SIZE + threadIdx.x;

    double sum = 0.0;

    // Обход всех "подблоков" A и B по горизонтали и вертикали соответственно
    for (int i = 0; i < (M + BLOCK_SIZE - 1) / BLOCK_SIZE; ++i) 
    {
        // Загрузка элементов в shared memory
        if (row < N && i * BLOCK_SIZE + threadIdx.x < M) 
        {
            tileA[threadIdx.y][threadIdx.x] = A[row * M + i * BLOCK_SIZE + threadIdx.x];
        }
        else {
            tileA[threadIdx.y][threadIdx.x] = 0.0;
        }

        if (col < K && i * BLOCK_SIZE + threadIdx.y < M) 
        {
            tileB[threadIdx.y][threadIdx.x] = B[(i * BLOCK_SIZE + threadIdx.y) * K + col];
        }
        else {
            tileB[threadIdx.y][threadIdx.x] = 0.0;
        }

        __syncthreads();

        // Умножение текущих подблоков A и B
        for (int j = 0; j < BLOCK_SIZE; ++j) 
        {
            sum += tileA[threadIdx.y][j] * tileB[j][threadIdx.x];
        }

        __syncthreads();
    }

    // Запись результата в глобальную память
    if (row < N && col < K) {
        C[row * K + col] = sum;
    }
}

void run_dgemmCUDAv2(const double* d_A, const double* d_B, double* d_C, int N, int M, int K)
{
    dim3 dimBlock(BLOCK_SIZE, BLOCK_SIZE);
    dim3 dimGrid((K + BLOCK_SIZE - 1) / BLOCK_SIZE, (N + BLOCK_SIZE - 1) / BLOCK_SIZE);

    blas_dgemmCUDAv2 << <dimGrid, dimBlock >> > (d_A, d_B, d_C, N, M, K);
}

void transpose_matrix(const double* B, double* B_transposed, int M, int K) 
{
    for (int i = 0; i < M; ++i) 
    {
        for (int j = 0; j < K; ++j) 
        {
            B_transposed[j * M + i] = B[i * K + j];
        }
    }
}

int main(int argc, char** argv)
{
    /* Declaration variables of matrix sizes */
    int N, M, K;

    /* Checking the number of command line arguments */
    if (argc == 2)
    {
        /* Initialization of matrix sizes */
        N = M = K = atoi(argv[0]);
    }
    else if (argc == 1)
    {
        N = M = K = 4096;
    }

    /* Allocation of memory for arrays A, B and result matrix C (host)*/
    double* A = (double*)malloc(N * M * sizeof(double));
    double* B = (double*)malloc(M * K * sizeof(double));
    double* C = (double*)malloc(N * K * sizeof(double));

    double* B_transposed = (double*)malloc(M * K * sizeof(double));
    transpose_matrix(B, B_transposed, M, K);

    if (!A || !B || !C)
    {
        fprintf(stderr, "Memory allocation failed\n");
        return 1;
    }

    init_matrix(N, M, K, A, B, C);

    /* Allocation of memory for arrays A, B and result matrix C (device)*/
    double* d_A, * d_B, * d_C;

    hipMalloc((void**)&d_A, N * M * sizeof(double));
    hipMalloc((void**)&d_B, M * K * sizeof(double));
    hipMalloc((void**)&d_C, N * K * sizeof(double));

    /* Copy arrays A, B and C out host memory in device memory*/
    hipMemcpy(d_A, A, N * M * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(d_B, B_transposed, M * K * sizeof(double), hipMemcpyHostToDevice);

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start);

    run_dgemmCUDAv2(d_A, d_B, d_C, N, M, K);

    hipEventRecord(stop);

    hipMemcpy(C, d_C, N * K * sizeof(double), hipMemcpyDeviceToHost);

    hipEventSynchronize(stop);
    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);
    printf("Matrix multiplication took %f milliseconds.\n", milliseconds);

    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);

    //print_matrix(N, K, C);

    /* Free allocated memory */
    free(A);
    free(B);
    free(C);

    return 0;
}

/* Function for displaying the matrix to the console */
void print_matrix(int N, int M, double* matrix)
{
    int i, j;
    for (i = 0; i < N; i++)
    {
        for (j = 0; j < M; j++)
        {
            printf("%f\t", matrix[i * N + j]);
        }
        printf("\n");
    }
}

/* The initialization function of matrices A, B and C */
void init_matrix(int N, int M, int K, double* A, double* B, double* C)
{
    int i, j;

    /* Initialization of matrix A (N x M) */
    for (i = 0; i < N; i++)
    {
        for (j = 0; j < M; j++)
        {
            A[i * M + j] = 1.0;
        }
    }

    /* Initialization of matrix B (M x K) */
    for (i = 0; i < M; i++)
    {
        for (j = 0; j < K; j++)
        {
            B[i * K + j] = 1.0;
        }
    }

    /* Initialization of matrix C (N x K) */
    for (i = 0; i < N; i++)
    {
        for (j = 0; j < K; j++)
        {
            C[i * K + j] = 0.0;
        }
    }
}

